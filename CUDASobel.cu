
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define THRESH 100

typedef struct {
    unsigned char *data; 
    int Width;           
    int Height;          
} Image;

// Function to create a new image
Image *CreateNewImage(int width, int height) {
    Image *image = (Image *)malloc(sizeof(Image));
    if (image == NULL) {
        printf("Error: Memory allocation failed\n");
        exit(EXIT_FAILURE);
    }
    image->Width = width;
    image->Height = height;
    image->data = (unsigned char *)malloc(width * height * sizeof(unsigned char));
    if (image->data == NULL) {
        printf("Error: Memory allocation failed\n");
        exit(EXIT_FAILURE);
    }
    return image;
}

// Function to free image data
void FreeImage(Image *image) {
    free(image->data);
    free(image);
}

// Function to convert RGB to grayscale
void rgb2gray(Image *image, unsigned char *rgbData) {
    for (int i = 0; i < image->Width * image->Height; ++i) {
        // Convert RGB to grayscale using luminosity method
        unsigned char gray = (unsigned char)(0.21 * rgbData[i * 3] + 0.72 * rgbData[i * 3 + 1] + 0.07 * rgbData[i * 3 + 2]);
        image->data[i] = gray;
    }
}

// Function to load an image from file
Image *LoadImage(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        printf("Error: Unable to open file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    char magic_number[3];
    fscanf(file, "%2s", magic_number);
    int width, height, max_value;
    fscanf(file, "%d %d %d", &width, &height, &max_value);
    fgetc(file); // Consume newline

    unsigned char *rgbData;
    if (magic_number[1] == '6') { // PPM binary format
        rgbData = (unsigned char *)malloc(width * height * 3 * sizeof(unsigned char));
        if (!rgbData) {
            printf("Error: Memory allocation failed\n");
            exit(EXIT_FAILURE);
        }
        fread(rgbData, sizeof(unsigned char), width * height * 3, file);
    } else if (magic_number[1] == '3') { // PPM ASCII format
        rgbData = (unsigned char *)malloc(width * height * 3 * sizeof(unsigned char));
        if (!rgbData) {
            printf("Error: Memory allocation failed\n");
            exit(EXIT_FAILURE);
        }
        for (int i = 0; i < width * height * 3; ++i) {
            fscanf(file, "%hhu", &rgbData[i]);
            fgetc(file); 
        }
    } else {
        printf("Error: Unsupported image format\n");
        exit(EXIT_FAILURE);
    }

    fclose(file);

    Image *image = CreateNewImage(width, height);
    rgb2gray(image, rgbData);

    free(rgbData);

    return image;
}

// Function to save an image to file in PGM format
void SaveImage(const char *filename, Image *image) {
    FILE *file = fopen(filename, "wb");
    if (!file) {
        printf("Error: Unable to open file %s for writing\n", filename);
        exit(EXIT_FAILURE);
    }

    fprintf(file, "P5\n%d %d\n255\n", image->Width, image->Height);
    fwrite(image->data, 1, image->Width * image->Height, file);

    fclose(file);
}

__global__ void optimized_kernel(unsigned char *d_in, unsigned char *d_out, int width, int height, int widthStep) {
    __shared__ unsigned char tile[18][18]; 

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int local_col = threadIdx.x + 1;
    int local_row = threadIdx.y + 1;

    if (col < width && row < height) {
        tile[local_row][local_col] = d_in[row * widthStep + col];
    }
    __syncthreads();

    if (col < width && row < height) {
        int dx[3][3] = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };
        int dy[3][3] = { 1, 2, 1, 0, 0, 0, -1, -2, -1 };

        int sum_x = 0;
        int sum_y = 0;

        for (int m = -1; m <= 1; m++) {
            for (int n = -1; n <= 1; n++) {
                sum_x += tile[local_row + m][local_col + n] * dx[m + 1][n + 1];
                sum_y += tile[local_row + m][local_col + n] * dy[m + 1][n + 1];
            }
        }

        int sum = abs(sum_x) + abs(sum_y);
        sum = min(255, sum); 

        d_out[row * widthStep + col] = sum;
    }
}

int main() {
    Image *input_image = LoadImage("flowers.ppm");
    int width = input_image->Width;
    int height = input_image->Height;

    Image *output_image = CreateNewImage(width, height);

    unsigned char *d_in, *d_out;
    size_t imageSize = width * height * sizeof(unsigned char);
    hipMalloc((void**)&d_in, imageSize);
    hipMalloc((void**)&d_out, imageSize);

    hipMemcpy(d_in, input_image->data, imageSize, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    clock_t start = clock();
    optimized_kernel<<<grid, block>>>(d_in, d_out, width, height, width);
    hipDeviceSynchronize(); 
    clock_t end = clock();

    hipMemcpy(output_image->data, d_out, imageSize, hipMemcpyDeviceToHost);

    SaveImage("sobelGPU_optimized2.pgm", output_image);

    hipFree(d_in);
    hipFree(d_out);
    FreeImage(input_image);
    FreeImage(output_image);

    double time_taken = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("Execution Time: %f seconds\n", time_taken);

    return 0;
}
